#include <iostream>
#include <vector>
#include <algorithm>
#include <cstdlib>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <random>

#define CUDA_CHECK(err) if (err != hipSuccess) { \
    std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl; \
    exit(EXIT_FAILURE); \
}

// Función para inicializar el array con números aleatorios usando std::mt19937
void initialize_array(std::vector<int>& array) {
    std::mt19937 gen(std::random_device{}());
    std::uniform_int_distribution<int> dist(0, 1000000);
    for (auto& val : array)
        val = dist(gen);
}

// Merge Sort paralelo usando OpenMP
void merge(std::vector<int>& array, int left, int mid, int right) {
    std::vector<int> temp(right - left + 1);
    int i = left, j = mid + 1, k = 0;

    while (i <= mid && j <= right)
        temp[k++] = (array[i] <= array[j]) ? array[i++] : array[j++];

    while (i <= mid) temp[k++] = array[i++];
    while (j <= right) temp[k++] = array[j++];

    for (int l = left; l <= right; ++l)
        array[l] = temp[l - left];
}

void merge_sort_parallel(std::vector<int>& array, int left, int right, int depth) {
    if (left >= right) return;

    int mid = left + (right - left) / 2;

    if (depth > 0) {
        #pragma omp parallel sections
        {
            #pragma omp section
            merge_sort_parallel(array, left, mid, depth - 1);
            #pragma omp section
            merge_sort_parallel(array, mid + 1, right, depth - 1);
        }
    } else {
        merge_sort_parallel(array, left, mid, 0);
        merge_sort_parallel(array, mid + 1, right, 0);
    }

    merge(array, left, mid, right);
}

void sort_cpu(std::vector<int>& array, int num_threads) {
    omp_set_num_threads(num_threads);
    merge_sort_parallel(array, 0, array.size() - 1, omp_get_max_active_levels());
}

// Ordenamiento en GPU usando Thrust
void sort_gpu(std::vector<int>& array) {
    thrust::device_vector<int> d_array(array.begin(), array.end());
    thrust::sort(d_array.begin(), d_array.end());
    thrust::copy(d_array.begin(), d_array.end(), array.begin());
}

int main(int argc, char *argv[]) {
    if (argc < 4) {
        std::cerr << "Uso: ./prog <n> <modo> <nt>\n";
        return EXIT_FAILURE;
    }

    size_t n = std::stoull(argv[1]); // Tamaño del array
    int modo = std::stoi(argv[2]);  // Modo: 0 = CPU, 1 = GPU
    int nt = std::stoi(argv[3]);    // Número de threads para CPU

    // Crear e inicializar el array
    std::vector<int> array(n);
    initialize_array(array);

    double start, end;

    if (modo == 0) { // CPU
        start = omp_get_wtime();
        sort_cpu(array, nt);
        end = omp_get_wtime();
    } else if (modo == 1) { // GPU
        start = omp_get_wtime();
        sort_gpu(array);
        end = omp_get_wtime();
    } else {
        std::cerr << "Modo no válido: 0 (CPU), 1 (GPU)\n";
        return EXIT_FAILURE;
    }

    std::cout << "Tiempo: " << (end - start) << " segundos\n";
    return 0;
}
