
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <random>
#include <chrono>

using namespace std;

// Función para inicializar el array con números aleatorios
void initialize_array(vector<int>& array) {
    random_device rd;
    mt19937 gen(rd());
    uniform_int_distribution<int> dist(0, 1000000);
    for (auto& val : array)
        val = dist(gen);
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        cerr << "Uso: ./prog <n>\n";
        return EXIT_FAILURE;
    }

    // Tamaño del array
    size_t n = stoull(argv[1]);

    // Crear e inicializar el array
    vector<int> array(n);
    initialize_array(array);

    // Ordenar usando std::sort y medir el tiempo
    auto start = chrono::high_resolution_clock::now();
    std::sort(array.begin(), array.end());
    auto end = chrono::high_resolution_clock::now();

    chrono::duration<double> elapsed = end - start;

    cout << "Tiempo std::sort: " << elapsed.count() << " segundos\n";

    return 0;
}
